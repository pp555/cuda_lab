#include <hip/hip_runtime.h>


#include <cstdio>
#include <iostream>
#include <cstdlib>


#define BLOCK_SIZE 64
#define N 64

using namespace std;

void displayLastError(const string &msg)
{
	cout << "Last Error (" << msg << "):\t" << hipGetErrorString(hipGetLastError()) << endl;
}

__global__ void bSearchCuda(float *array, float search, int *index)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
    if(array[x]==search)
        *index = x;
}


int main(int argc, char *argv[])
{

    float *data = new float[N];
    for(int i=0;i<N;i++)
        data[i] = i;

    float *deviceData;
    int *deviceIndex;
    size_t size = N*sizeof(float);
    hipMalloc((void**)&deviceData, size);
	displayLastError("memory allocation");
    hipMalloc((void**)&deviceIndex, sizeof(int));
	displayLastError("memory allocation");

	hipMemcpy(deviceData, data, size, hipMemcpyHostToDevice);
	displayLastError("memory copying");

    int index = -1;
	hipMemcpy(deviceIndex, &index, sizeof(int), hipMemcpyHostToDevice);




    int blocks = N / BLOCK_SIZE;
    if(N % BLOCK_SIZE)
		blocks++;

    bSearchCuda<<<blocks, BLOCK_SIZE>>>(deviceData, 100.0f, deviceIndex);
	displayLastError("kernel");


	hipMemcpy(&index, deviceIndex, sizeof(int), hipMemcpyDeviceToHost);
	displayLastError("memory copying");

    cout << index << endl;


    hipFree(deviceData);
	displayLastError("free");
    hipFree(deviceIndex);
	displayLastError("free");
    delete [] data;

    return 0;

}

