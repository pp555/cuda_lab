#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <ctime>
#include <unistd.h>

using namespace std;

#define  N   		1000000
#define  BLOCK_SIZE	16
#define TIME_CHECK clock()/float(CLOCKS_PER_SEC)

float 	   hArray[N];
float     *dArray;
int 	   blocks;


void prologue(void) {
	memset(hArray, 0, sizeof(hArray));
	for(int i = 0; i < N; i++) {
		hArray[i] =  i + 1;
	}		
   	hipMalloc((void**)&dArray, sizeof(hArray));
   	hipMemcpy(dArray, hArray, sizeof(hArray), hipMemcpyHostToDevice);
}

void epilogue(void) {
	hipMemcpy(hArray, dArray, sizeof(hArray), hipMemcpyDeviceToHost);
	hipFree(dArray);
}


// Kernel
__global__ void pow3(float *A) {
	int x = blockDim.x * blockIdx.x + threadIdx.x; 

    if(x < N)
	    A[x] = A[x] * A[x] * A[x]; 
}

int main(int argc, char** argv)
{
    int	 devCnt;

    hipGetDeviceCount(&devCnt);
    if(devCnt == 0) {
		perror("No CUDA devices available -- exiting.");
		return 1;
    }



	float gpu_start_time = 0;
	float gpu_post_prologue_time = 0;
	float gpu_post_computing_time = 0;
	float gpu_end_time = 0;
	float cpu_start_time = 0;
	float cpu_end_time = 0;

	gpu_start_time = TIME_CHECK;

    prologue();
    blocks = N / BLOCK_SIZE;
    if(N % BLOCK_SIZE)
		blocks++;

	gpu_post_prologue_time = TIME_CHECK;

    pow3<<<blocks, BLOCK_SIZE>>>(dArray);
    hipDeviceSynchronize();

	gpu_post_computing_time = TIME_CHECK;

    epilogue();

	gpu_end_time = TIME_CHECK;


sleep(1);

//cpu
	cpu_start_time = TIME_CHECK;
	for(long long i=0;i<N;i++)
		hArray[i] = hArray[i] * hArray[i] * hArray[i];
	cpu_end_time = TIME_CHECK;


	cout << gpu_start_time << endl;
	cout << gpu_post_prologue_time << endl;
	cout << gpu_post_computing_time << endl;
	cout << gpu_end_time << endl;

	cout << cpu_start_time << endl;
	cout << cpu_end_time << endl;

	cout << "prologue\t" <<  gpu_post_prologue_time - gpu_start_time << endl;
	cout << "counting\t" <<  gpu_post_computing_time - gpu_post_prologue_time << endl;
	cout << "epilogue\t" <<  gpu_end_time - gpu_post_computing_time << endl;


	cout << "cpu\t" <<  cpu_end_time - cpu_start_time << endl;
    

    return 0;
}
